#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/>

#define CHECK_ERROR(call) {\
    const hipError_t err = call;\
    if (err != hipSuccess)\
    {\
        printf("Error:%s,%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",err,hipGetErrorString(err));\
        exit(1);\
    }\
}

__global__ void mirror_kernel(cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSz<uchar3> dst, int h, int w)
{
        unsigned int x;
        unsigned int y;
}